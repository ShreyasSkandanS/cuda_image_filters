#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

const int BLOCKDIM = 32;
const int MAX_WINDOW = 11;
__device__ const int FILTER_SIZE = 9;
__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_quick(float *x, int left_idx, int right_idx) 
{
      int i = left_idx, j = right_idx;
      float pivot = x[(left_idx + right_idx) / 2];
      while (i <= j) 
      {
            while (x[i] < pivot)
                  i++;
            while (x[j] > pivot)
                  j--;
            if (i <= j) {
		  float temp;
                  temp = x[i];
                  x[i] = x[j];
                  x[j] = temp;
                  i++;
                  j--;
            }
      };
      if (left_idx < j)
            sort_quick(x, left_idx, j);
      if (i < right_idx)
            sort_quick(x, i, right_idx);
}

__device__ void sort_bubble(float *x, int n_size) 
{
	for (int i = 0; i < n_size - 1; i++) 
	{
		for(int j = 0; j < n_size - i - 1; j++) 
		{
			if (x[j] > x[j+1]) 
			{
				float temp = x[j];
				x[j] = x[j+1];
				x[j+1] = temp;
			}
		}
	}
}

__device__ void sort_linear(float *x, int n_size) 
{
	for (int i = 0; i < n_size-1; i++) 
	{
		int min_idx = i;
		for (int j = i + 1; j < n_size; j++) 
		{
			if(x[j] < x[min_idx])
				min_idx = j;
		}
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}
}


__device__ int index(int x, int y, int width) 
{
	return (y * width) + x;
}

__device__ int clamp(int value, int bound) 
{
	if (value < 0) {
		return 1;
	}
	if (value < bound) {
		return value;
	}
	return bound - 1;
}

__global__ void median_filter_2d(unsigned char* input, unsigned char* output, int width, int height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x<width) && (y<height))
	{
		const int color_tid = index(x,y,width);
		float windowMedian[MAX_WINDOW*MAX_WINDOW];
		int windowElements = 0;
		for (int x_iter = x - FILTER_HALFSIZE; x_iter <= x + FILTER_HALFSIZE; x_iter ++)
		 {
			for (int y_iter = y - FILTER_HALFSIZE; y_iter <= y + FILTER_HALFSIZE; y_iter++)
			 {
				if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height)
				{
					windowMedian[windowElements++] = input[index(x_iter,y_iter,width)];
				}
			}
		}
		sort_bubble(windowMedian,windowElements);
		//sort_linear(windowMedian,windowElements);
		//sort_quick(windowMedian,0,windowElements);
		output[color_tid] = windowMedian[windowElements/2];
	}
}

__global__ void median_filter_2d_sm(unsigned char* input, unsigned char* output, int width, int height)
{
	__shared__ int sharedPixels[BLOCKDIM + FILTER_SIZE][BLOCKDIM + FILTER_SIZE];
	
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	int xBlockLimit_max = blockDim.x - FILTER_HALFSIZE - 1;
	int yBlockLimit_max = blockDim.y - FILTER_HALFSIZE - 1;
	int xBlockLimit_min = FILTER_HALFSIZE;
	int yBlockLimit_min = FILTER_HALFSIZE;

	if (threadIdx.x > xBlockLimit_max && threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
	    	unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x > xBlockLimit_max && threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min && threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min && threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y + FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x > xBlockLimit_max) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y] = pixel;
	}
	if (threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x,width), clamp(y + FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	int i = index(x, y, width);
	unsigned int pixel = input[i];
	sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;

	__syncthreads();

	if((x<width) && (y<height))
	{
		const int color_tid = y * width + x;
		float windowMedian[MAX_WINDOW*MAX_WINDOW];
		int windowElements = 0;

		for (int x_iter = 0; x_iter < FILTER_SIZE; x_iter ++) 
		{
			for (int y_iter = 0; y_iter < FILTER_SIZE; y_iter++) 
			{
				if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height) 
				{
					windowMedian[windowElements++] = sharedPixels[threadIdx.x + x_iter][threadIdx.y + y_iter];
				}
			}
		}
		sort_bubble(windowMedian,windowElements);
		//sort_linear(windowMedian,windowElements);
		//sort_quick(windowMedian,0,windowElements);
		output[color_tid] = windowMedian[windowElements/2];
	}
}


void median_filter_wrapper(const cv::Mat& input, cv::Mat& output)
{
	unsigned char *d_input, *d_output;
	
	hipError_t cudaStatus;	
	
	cudaStatus = hipMalloc<unsigned char>(&d_input,input.rows*input.cols);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipMalloc<unsigned char>(&d_output,output.rows*output.cols);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(d_input,input.ptr(),input.rows*input.cols,hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);	
	
	const dim3 block(BLOCKDIM,BLOCKDIM);
	const dim3 grid(input.cols/BLOCKDIM, input.rows/BLOCKDIM);

	median_filter_2d<<<grid,block>>>(d_input,d_output,input.cols,input.rows);

	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipMemcpy(output.ptr(),d_output,output.rows*output.cols,hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipFree(d_input);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipFree(d_output);
	checkCudaErrors(cudaStatus);	
}


